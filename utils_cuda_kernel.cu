#include <torch/extension.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <ATen/ATen.h>

void float2bin_cuda(
    torch::Tensor from,
    torch::Tensor to) {
    try{
//        float* pp=from.data_ptr<float>();
//        printf("ptr:%p",pp);
//        printf("numel:%d",from.numel());
//        auto shape=from.sizes();
//        auto output=torch::from_blob(ptr,shape,torch::TensorOptions().dtype(torch::kInt));
//        printf("size of float:%d,int:%d,long:%d",sizeof(float),sizeof(int),sizeof(long));
        auto hipError_t=hipMemcpy(to.data_ptr<int>(),from.data_ptr<float>(),sizeof(float)*from.numel(),hipMemcpyKind(3));
        if (hipError_t!=0) {
            printf("Error in hipMemcpy: %d",hipError_t);
        }
        return;
    }catch(const std::exception& e){
        printf("%s\n", e.what());
    }
    return;
}

void bin2float_cuda(
    torch::Tensor from,
    torch::Tensor to) {
    try{
        auto hipError_t=hipMemcpy(to.data_ptr<float>(),from.data_ptr<int>(),sizeof(float)*from.numel(),hipMemcpyKind(3));
        if (hipError_t!=0) {
            printf("Error in hipMemcpy: %d",hipError_t);
        }
        return;
    }catch(const std::exception& e){
        printf("%s\n", e.what());
    }
    return;
}